#include "hip/hip_runtime.h"
// File: AudioFilter.cu
// Compile with:
//   nvcc -std=c++17 AudioFilter.cu -I/path/to/AudioFile -o bin/AudioFilter

#define DR_WAV_IMPLEMENTATION
#include "AudioFile.h"
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <filesystem>

namespace fs = std::filesystem;


// CUDA error-check helper
inline void checkCuda(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg
                  << " (" << hipGetErrorString(err) << ")\n";
        std::exit(EXIT_FAILURE);
    }
}

// GPU kernel: low-pass FIR on interleaved data (supports N channels)
__global__
void lowPassInterleaved(const float* __restrict__ in,
                        float*       __restrict__ out,
                        int frames,
                        int channels,
                        int kernelRadius)
{
    int tid   = blockIdx.x * blockDim.x + threadIdx.x;
    int total = frames * channels;
    if (tid >= total) return;

    int frame = tid / channels;
    int ch    = tid % channels;

    float sum = 0.0f;
    int count = 0;
    for (int k = -kernelRadius; k <= kernelRadius; ++k) {
        int f = frame + k;
        if      (f <  0)      f = 0;
        else if (f >= frames) f = frames - 1;
        sum += in[f*channels + ch];
        ++count;
    }
    out[frame*channels + ch] = sum / float(count);
}


// Process a single WAV file: load, filter, save
void processFile(const fs::path &inPath, const fs::path &outPath) {
    std::cout << "Loading: " << inPath << "\n";
    AudioFile<float> audioFile;
    if (!audioFile.load(inPath.string())) {
        std::cerr << "ERROR: failed to load " << inPath << "\n";
        return;
    }

    int channels   = audioFile.getNumChannels();
    int frames     = audioFile.getNumSamplesPerChannel();
    int bitDepth   = audioFile.getBitDepth();
    float rate     = audioFile.getSampleRate();
    int samples    = frames * channels;

    std::cout << " Loaded " << samples << " samples @ "
              << rate << " Hz, " << bitDepth
              << "-bit, " << channels << "-channel\n";

    // Flatten interleaved host buffer
    std::vector<float> h_in(samples), h_out(samples);
    for (int ch = 0; ch < channels; ++ch) {
        const auto &chan = audioFile.samples[ch];
        for (int f = 0; f < frames; ++f) {
            h_in[f*channels + ch] = chan[f];
        }
    }

    // Allocate & copy to GPU
    float *d_in = nullptr, *d_out = nullptr;
    size_t bytes = size_t(samples) * sizeof(float);
    checkCuda(hipMalloc(&d_in,  bytes), "hipMalloc d_in");
    checkCuda(hipMalloc(&d_out, bytes), "hipMalloc d_out");
    checkCuda(hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice),
              "H2D memcpy");
    std::cout << " Copied to GPU\n";

    // Launch kernel
    const int KERNEL_RADIUS = 16;   // radius -> taps = 2*radius+1
    int threads = 256;
    int blocks  = (samples + threads - 1) / threads;
    lowPassInterleaved<<<blocks, threads>>>(
        d_in, d_out, frames, channels, KERNEL_RADIUS
    );
    checkCuda(hipPeekAtLastError(),  "kernel launch");
    checkCuda(hipDeviceSynchronize(), "kernel sync");
    std::cout << " Low-pass FIR done (radius=" << KERNEL_RADIUS << ")\n";

    // Copy back & un-flatten
    checkCuda(hipMemcpy(h_out.data(), d_out, bytes, hipMemcpyDeviceToHost),
              "D2H memcpy");
    std::cout << " Copied back to host\n";

    AudioFile<float> outFile;
    outFile.setAudioBufferSize(channels, frames);
    outFile.setSampleRate(rate);
    outFile.setBitDepth(bitDepth);
    for (int ch = 0; ch < channels; ++ch) {
        for (int f = 0; f < frames; ++f) {
            outFile.samples[ch][f] = h_out[f*channels + ch];
        }
    }

    if (outFile.save(outPath.string())) {
        std::cout << " Saved filtered WAV to " << outPath << "\n";
    } else {
        std::cerr << "ERROR: failed to save " << outPath << "\n";
    }

    hipFree(d_in);
    hipFree(d_out);
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0]
                  << " <input_dir> <output_dir>\n";
        return 1;
    }

    fs::path inDir  = argv[1];
    fs::path outDir = argv[2];

    // Create output directory
    fs::create_directories(outDir);

    for (auto &entry : fs::directory_iterator(inDir)) {
        if (!entry.is_regular_file()) continue;
        if (entry.path().extension() != ".wav") continue;

        fs::path inPath  = entry.path();
        std::string stem = inPath.stem().string();
        fs::path outPath = outDir / (stem + "_filtered.wav");

        processFile(inPath, outPath);
    }

    std::cout << "Done.\n";
    return 0;
}
